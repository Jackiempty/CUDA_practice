#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <iomanip>
#include <iostream>

#include "naive.cuh"
#include "optimized.cuh"

#define M 256  // Number of rows in A and C
#define N 512  // Number of columns in A and rows in B
#define K 256  // Number of columns in B and C
#define BLOCK_SIZE 32
using namespace std;

// Example 3x2 @ 2x4 = 3x4 -> (M x K) @ (K x N) = (M x N)
// A = [[1, 2],
//      [3, 4],
//      [5, 6]]

// B = [[7, 8, 9, 10],
//      [11, 12, 13, 14]]

// C = A * B = [[1*7 + 2*11, 1*8 + 2*12, 1*9 + 2*13, 1*10 + 2*14],
//              [3*7 + 4*11, 3*8 + 4*12, 3*9 + 4*13, 3*10 + 4*14],
//              [5*7 + 6*11, 5*8 + 6*12, 5*9 + 6*13, 5*10 + 6*14]]

// C = [[29, 32, 35, 38],
//      [65, 72, 79, 86],
//      [101, 112, 123, 134]]

// Function to measure execution time
double get_time() {
    struct timespec ts;
    clock_gettime(CLOCK_MONOTONIC, &ts);
    return ts.tv_sec + ts.tv_nsec * 1e-9;
}

// CPU matrix multiplication
void matmul_cpu(float* A, float* B, float* C, int m, int n, int k) {
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < k; j++) {
            float sum = 0.0f;
            for (int l = 0; l < n; l++) {
                sum += A[i * n + l] * B[l * k + j];
            }
            C[i * k + j] = sum;
        }
    }
}

// Initialize matrix with random values
void init_matrix(float* mat, int rows, int cols) {
    for (int i = 0; i < rows * cols; i++) {
        mat[i] = (float)rand() / RAND_MAX;
    }
}

int main() {
    float *h_a, *h_b, *h_c_cpu, *h_c_gpu_naive, *h_c_gpu_optimized;
    float *d_a, *d_b, *d_c_naive, *d_c_optimized;
    int size_A = M * N * sizeof(float);
    int size_B = N * K * sizeof(float);
    int size_C = M * K * sizeof(float);

    // Allocate host memory
    h_a = (float*)malloc(size_A);
    h_b = (float*)malloc(size_B);
    h_c_cpu = (float*)malloc(size_C);
    h_c_gpu_naive = (float*)malloc(size_C);
    h_c_gpu_optimized = (float*)malloc(size_C);

    // Initialize matrices
    srand(time(NULL));
    init_matrix(h_a, M, N);
    init_matrix(h_b, N, K);

    // Allocate device memory
    hipMalloc(&d_a, size_A);
    hipMalloc(&d_b, size_B);
    hipMalloc(&d_c_naive, size_C);
    hipMalloc(&d_c_optimized, size_C);

    // Copy data to device
    hipMemcpy(d_a, h_a, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size_B, hipMemcpyHostToDevice);

    // Warm-up runs
    printf("Performing warm-up runs...\n");
    for (int i = 0; i < 3; i++) {
        // vector_add_cpu(h_a, h_b, h_c_cpu, N);
        matmul_cpu(h_a, h_b, h_c_cpu, M, N, K);
    }

    // Benchmark CPU implementation
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0.0f;

    printf("Benchmarking CPU implementation...\n");
    double cpu_total_time = 0.0;
    for (int i = 0; i < 5; i++) {
        double start_time = get_time();
        matmul_cpu(h_a, h_b, h_c_cpu, M, N, K);
        double end_time = get_time();
        cpu_total_time += end_time - start_time;
    }
    double cpu_avg_time = cpu_total_time / 5.0;

    // Benchmark GPU naive implementation
    printf("Benchmarking GPU naive implementation...\n");
    double gpu_naive_total_time = 0.0;
    double gpu_naive_total_cuda_time = 0.0;
    for (int i = 0; i < 100; i++) {
        hipMemset(d_c_naive, 0, size_C);  // Clear previous results
        double start_time = get_time();
        naive(d_a, d_b, d_c_naive, M, N, K, &start, &stop);
        double end_time = get_time();
        gpu_naive_total_time += end_time - start_time;
        // cuda event time
        milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        gpu_naive_total_cuda_time += milliseconds;
    }
    double gpu_naive_avg_time = gpu_naive_total_time / 100.0;
    double gpu_naive_avg_cuda_time = gpu_naive_total_cuda_time / 100.0;

    // Verify naive results immediately
    hipMemcpy(h_c_gpu_naive, d_c_naive, size_C, hipMemcpyDeviceToHost);
    bool correct_naive = true;
    for (int i = 0; i < N; i++) {
        if (fabs(h_c_cpu[i] - h_c_gpu_naive[i]) > 1e-4) {
            correct_naive = false;
            cout << i << " cpu: " << h_c_cpu[i] << " != " << h_c_gpu_naive[i] << endl;
            break;
        }
    }
    printf("Naive Results are %s\n", correct_naive ? "correct" : "incorrect");

    // Benchmark GPU optimized implementation
    printf("Benchmarking GPU optimized implementation...\n");
    double gpu_optimized_total_time = 0.0;
    double gpu_optimized_total_cuda_time = 0.0;
    for (int i = 0; i < 100; i++) {
        hipMemset(d_c_optimized, 0, size_C);  // Clear previous results
        double start_time = get_time();
        optimized(d_a, d_b, d_c_optimized, M, N, K, &start, &stop);
        double end_time = get_time();
        gpu_optimized_total_time += end_time - start_time;
        // cuda event time
        milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        gpu_optimized_total_cuda_time += milliseconds;
    }
    double gpu_optimized_avg_time = gpu_optimized_total_time / 100.0;
    double gpu_optimized_avg_cuda_time = gpu_optimized_total_cuda_time / 100.0;

    // Verify optimized results immediately
    hipMemcpy(h_c_gpu_optimized, d_c_optimized, size_C, hipMemcpyDeviceToHost);
    bool correct_optimized = true;
    for (int i = 0; i < N; i++) {
        if (fabs(h_c_cpu[i] - h_c_gpu_optimized[i]) > 1e-4) {
            correct_optimized = false;
            cout << i << " cpu: " << h_c_cpu[i] << " != " << h_c_gpu_optimized[i] << endl;
            break;
        }
    }
    printf("Optimized Results are %s\n", correct_optimized ? "correct" : "incorrect");

    // Print results
    cout << endl;
    cout << left << setw(35) << "CPU average time:" << right << setw(12) << fixed << setprecision(3)
         << cpu_avg_time * 1000 << " ms" << endl;
    cout << left << setw(35) << "GPU naive average time:" << right << setw(12) << fixed << setprecision(3)
         << gpu_naive_avg_time * 1000 << " ms" << endl;
    cout << left << setw(35) << "GPU naive average cuda time:" << right << setw(12) << fixed << setprecision(3)
         << gpu_naive_avg_cuda_time << " ms" << endl;
    cout << left << setw(35) << "GPU optimized average time:" << right << setw(12) << fixed << setprecision(3)
         << gpu_optimized_avg_time * 1000 << " ms" << endl;
    cout << left << setw(35) << "GPU optimized average cuda time:" << right << setw(12) << fixed << setprecision(3)
         << gpu_optimized_avg_cuda_time << " ms" << endl;
    cout << left << setw(35) << "Speedup (CPU vs GPU naive):" << right << setw(12) << fixed << setprecision(3)
         << cpu_avg_time / gpu_naive_avg_time << "x" << endl;
    cout << left << setw(35) << "Speedup (CPU vs GPU optimized):" << right << setw(12) << fixed << setprecision(3)
         << cpu_avg_time / gpu_optimized_avg_time << "x" << endl;
    cout << left << setw(35) << "Speedup (GPU naive vs GPU optimized):" << right << setw(12) << fixed << setprecision(3)
         << gpu_naive_avg_time / gpu_optimized_avg_time << "x" << endl;

    // destroy event
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Free memory
    free(h_a);
    free(h_b);
    free(h_c_cpu);
    free(h_c_gpu_naive);
    free(h_c_gpu_optimized);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c_naive);
    hipFree(d_c_optimized);

    return 0;
}