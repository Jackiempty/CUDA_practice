#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <iostream>

#include "naive.cuh"
#include "optimized.cuh"

#define N 10000000  // Vector size = 10 million

// Function to measure execution time
double get_time() {
    struct timespec ts;
    clock_gettime(CLOCK_MONOTONIC, &ts);
    return ts.tv_sec + ts.tv_nsec * 1e-9;
}

// CPU vector addition
void vector_add_cpu(float* a, float* b, float* c, int n) {
    for (int i = 0; i < n; i++) {
        c[i] = a[i] + b[i];
    }
}

// Initialize vector with random values
void init_vector(float* vec, int n) {
    for (int i = 0; i < n; i++) {
        vec[i] = (float)rand() / RAND_MAX;
    }
}

int main() {
    float *h_a, *h_b, *h_c_cpu, *h_c_gpu_naive, *h_c_gpu_optimized;
    float *d_a, *d_b, *d_c_naive, *d_c_optimized;
    size_t size = N * sizeof(float);

    // Allocate host memory
    h_a = (float*)malloc(size);
    h_b = (float*)malloc(size);
    h_c_cpu = (float*)malloc(size);
    h_c_gpu_naive = (float*)malloc(size);
    h_c_gpu_optimized = (float*)malloc(size);

    // Initialize vectors
    srand(time(NULL));
    init_vector(h_a, N);
    init_vector(h_b, N);

    // Allocate device memory
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c_naive, size);
    hipMalloc(&d_c_optimized, size);

    // Copy data to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    // Warm-up runs
    printf("Performing warm-up runs...\n");
    for (int i = 0; i < 3; i++) {
        vector_add_cpu(h_a, h_b, h_c_cpu, N);
        // naive(d_a, d_b, d_c_naive, N);
        // hipDeviceSynchronize();
    }

    // Benchmark CPU implementation
    printf("Benchmarking CPU implementation...\n");
    double cpu_total_time = 0.0;
    for (int i = 0; i < 5; i++) {
        double start_time = get_time();
        vector_add_cpu(h_a, h_b, h_c_cpu, N);
        double end_time = get_time();
        cpu_total_time += end_time - start_time;
    }
    double cpu_avg_time = cpu_total_time / 5.0;

    // Benchmark GPU naive implementation
    printf("Benchmarking GPU naive implementation...\n");
    double gpu_naive_total_time = 0.0;
    for (int i = 0; i < 100; i++) {
        hipMemset(d_c_naive, 0, size);  // Clear previous results
        double start_time = get_time();
        naive(d_a, d_b, d_c_naive, N);
        double end_time = get_time();
        gpu_naive_total_time += end_time - start_time;
    }
    double gpu_naive_avg_time = gpu_naive_total_time / 100.0;

    // Verify naive results immediately
    hipMemcpy(h_c_gpu_naive, d_c_naive, size, hipMemcpyDeviceToHost);
    bool correct_naive = true;
    for (int i = 0; i < N; i++) {
        if (fabs(h_c_cpu[i] - h_c_gpu_naive[i]) > 1e-4) {
            correct_naive = false;
            std::cout << i << " cpu: " << h_c_cpu[i] << " != " << h_c_gpu_naive[i] << std::endl;
            break;
        }
    }
    printf("Naive Results are %s\n", correct_naive ? "correct" : "incorrect");

    // Benchmark GPU optimized implementation
    printf("Benchmarking GPU optimized implementation...\n");
    double gpu_optimized_total_time = 0.0;
    for (int i = 0; i < 100; i++) {
        hipMemset(d_c_optimized, 0, size);  // Clear previous results
        double start_time = get_time();
        optimized(d_a, d_b, d_c_optimized, N);
        double end_time = get_time();
        gpu_optimized_total_time += end_time - start_time;
    }
    double gpu_optimized_avg_time = gpu_optimized_total_time / 100.0;

    // Verify optimized results immediately
    hipMemcpy(h_c_gpu_optimized, d_c_optimized, size, hipMemcpyDeviceToHost);
    bool correct_optimized = true;
    for (int i = 0; i < N; i++) {
        if (fabs(h_c_cpu[i] - h_c_gpu_optimized[i]) > 1e-4) {
            correct_optimized = false;
            std::cout << i << " cpu: " << h_c_cpu[i] << " != " << h_c_gpu_optimized[i] << std::endl;
            break;
        }
    }
    printf("Optimized Results are %s\n", correct_optimized ? "correct" : "incorrect");

    // Print results
    printf("CPU average time: %f milliseconds\n", cpu_avg_time * 1000);
    printf("GPU naive average time: %f milliseconds\n", gpu_naive_avg_time * 1000);
    printf("GPU optimized average time: %f milliseconds\n", gpu_optimized_avg_time * 1000);
    printf("Speedup (CPU vs GPU naive): %fx\n", cpu_avg_time / gpu_naive_avg_time);
    printf("Speedup (CPU vs GPU optimized): %fx\n", cpu_avg_time / gpu_optimized_avg_time);
    printf("Speedup (GPU naive vs GPU optimized): %fx\n", gpu_naive_avg_time / gpu_optimized_avg_time);

    // Free memory
    free(h_a);
    free(h_b);
    free(h_c_cpu);
    free(h_c_gpu_naive);
    free(h_c_gpu_optimized);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c_naive);
    hipFree(d_c_optimized);

    return 0;
}