#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <iostream>
#include <iomanip>

#include "naive.cuh"
#include "optimized.cuh"

#define M 256  // Number of rows in A and C
#define N 512  // Number of columns in A and rows in B
using namespace std;

// Function to measure execution time
double get_time() {
    struct timespec ts;
    clock_gettime(CLOCK_MONOTONIC, &ts);
    return ts.tv_sec + ts.tv_nsec * 1e-9;
}

// CPU matrix transpose
void mat_trans_cpu(float* input, float* output, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            output[j * rows + i] = input[i * cols + j];
        }
    }
}

// Initialize matrix with random values
void init_matrix(float* mat, int rows, int cols) {
    for (int i = 0; i < rows * cols; i++) {
        mat[i] = (float)rand() / RAND_MAX;
    }
}

int main() {
    float *h_input, *h_output_cpu, *h_output_gpu_naive, *h_output_gpu_optimized;
    float *d_input, *d_output_naive, *d_output_optimized;
    int size = M * N * sizeof(float);

    // Allocate host memory
    h_input = (float*)malloc(size);
    h_output_cpu = (float*)malloc(size);
    h_output_gpu_naive = (float*)malloc(size);
    h_output_gpu_optimized = (float*)malloc(size);

    // Initialize matrices
    srand(time(NULL));
    init_matrix(h_input, M, N);

    // Allocate device memory
    hipMalloc(&d_input, size);
    hipMalloc(&d_output_naive, size);
    hipMalloc(&d_output_optimized, size);

    // Copy data to device
    hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice);

    // Warm-up runs
    printf("Performing warm-up runs...\n");
    for (int i = 0; i < 3; i++) {
        // vector_add_cpu(h_a, h_b, h_c_cpu, N);
        mat_trans_cpu(h_input, h_output_cpu, M, N);
    }

    // Benchmark CPU implementation
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0.0f;

    printf("Benchmarking CPU implementation...\n");
    double cpu_total_time = 0.0;
    for (int i = 0; i < 5; i++) {
        double start_time = get_time();
        mat_trans_cpu(h_input, h_output_cpu, M, N);
        double end_time = get_time();
        cpu_total_time += end_time - start_time;
    }
    double cpu_avg_time = cpu_total_time / 5.0;

    // Benchmark GPU naive implementation
    printf("Benchmarking GPU naive implementation...\n");
    double gpu_naive_total_time = 0.0;
    double gpu_naive_total_cuda_time = 0.0;
    for (int i = 0; i < 100; i++) {
        hipMemset(d_output_naive, 0, size);  // Clear previous results
        double start_time = get_time();
        naive(d_input, d_output_naive, M, N, &start, &stop);
        double end_time = get_time();
        gpu_naive_total_time += end_time - start_time;
        // cuda event time
        milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        gpu_naive_total_cuda_time += milliseconds;
    }
    double gpu_naive_avg_time = gpu_naive_total_time / 100.0;
    double gpu_naive_avg_cuda_time = gpu_naive_total_cuda_time / 100.0;

    // Verify naive results immediately
    hipMemcpy(h_output_gpu_naive, d_output_naive, size, hipMemcpyDeviceToHost);
    bool correct_naive = true;
    for (int i = 0; i < N; i++) {
        if (fabs(h_output_cpu[i] - h_output_gpu_naive[i]) > 1e-4) {
            correct_naive = false;
            cout << i << " cpu: " << h_output_cpu[i] << " != " << h_output_gpu_naive[i] << endl;
            break;
        }
    }
    printf("Naive Results are %s\n", correct_naive ? "correct" : "incorrect");

    // Benchmark GPU optimized implementation
    printf("Benchmarking GPU optimized implementation...\n");
    double gpu_optimized_total_time = 0.0;
    double gpu_optimized_total_cuda_time = 0.0;
    for (int i = 0; i < 100; i++) {
        hipMemset(d_output_optimized, 0, size);  // Clear previous results
        double start_time = get_time();
        optimized(d_input, d_output_optimized, M, N, &start, &stop);
        double end_time = get_time();
        gpu_optimized_total_time += end_time - start_time;
        // cuda event time
        milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        gpu_optimized_total_cuda_time += milliseconds;
    }
    double gpu_optimized_avg_time = gpu_optimized_total_time / 100.0;
    double gpu_optimized_avg_cuda_time = gpu_optimized_total_cuda_time / 100.0;

    // Verify optimized results immediately
    hipMemcpy(h_output_gpu_optimized, d_output_optimized, size, hipMemcpyDeviceToHost);
    bool correct_optimized = true;
    for (int i = 0; i < N; i++) {
        if (fabs(h_output_cpu[i] - h_output_gpu_optimized[i]) > 1e-4) {
            correct_optimized = false;
            cout << i << " cpu: " << h_output_cpu[i] << " != " << h_output_gpu_optimized[i] << endl;
            break;
        }
    }
    printf("Optimized Results are %s\n", correct_optimized ? "correct" : "incorrect");

    // Print results
    cout << endl;
    cout << left << setw(35) << "CPU average time:" << right << setw(12) << fixed << setprecision(3)
         << cpu_avg_time * 1000 << " ms" << endl;
    cout << left << setw(35) << "GPU naive average time:" << right << setw(12) << fixed << setprecision(3)
         << gpu_naive_avg_time * 1000 << " ms" << endl;
    cout << left << setw(35) << "GPU naive average cuda time:" << right << setw(12) << fixed << setprecision(3)
         << gpu_naive_avg_cuda_time << " ms" << endl;
    cout << left << setw(35) << "GPU optimized average time:" << right << setw(12) << fixed << setprecision(3)
         << gpu_optimized_avg_time * 1000 << " ms" << endl;
    cout << left << setw(35) << "GPU optimized average cuda time:" << right << setw(12) << fixed << setprecision(3)
         << gpu_optimized_avg_cuda_time << " ms" << endl;
    cout << left << setw(35) << "Speedup (CPU vs GPU naive):" << right << setw(12) << fixed << setprecision(3)
         << cpu_avg_time / gpu_naive_avg_time << "x" << endl;
    cout << left << setw(35) << "Speedup (CPU vs GPU optimized):" << right << setw(12) << fixed << setprecision(3)
         << cpu_avg_time / gpu_optimized_avg_time << "x" << endl;
    cout << left << setw(35) << "Speedup (GPU naive vs GPU optimized):" << right << setw(12) << fixed << setprecision(3)
         << gpu_naive_avg_time / gpu_optimized_avg_time << "x" << endl;

    // destroy event
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    // Free memory
    free(h_input);
    free(h_output_cpu);
    free(h_output_gpu_naive);
    free(h_output_gpu_optimized);
    hipFree(d_input);
    hipFree(d_output_naive);
    hipFree(d_output_optimized);

    return 0;
}